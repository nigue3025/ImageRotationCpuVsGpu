#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
#include<vector>
#include<time.h>
//#define printMatrix
hipError_t rotationCuda(int* a, std::vector<std::vector<int>> matrix);




__global__ void showThreadNo()
{
	// int tid = blockDim.x * blockIdx.x + threadIdx.x;
	printf("idx %d, idy %d\n", threadIdx.x, threadIdx.y);// << std::endl;

}

__global__ void rotateKernel_badVersion(int matrixSz, int halfsz, int* matrix)
{
	// int tid = blockDim.x * blockIdx.x + threadIdx.x;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sz = matrixSz;

	//printf("(%d):%d\n", tid,matrix[tid]);

	// int halfsz = ceil((double)sz / 2.0);
	int tempx = 0, tempy = 0;
	int temp = 0, tempPrev = 0;


	for (int j = tid; j < sz - 1 - tid; j++)
	{
		tempx = tid;
		tempy = j;
		for (int l = 0; l < 4; l++)
		{
			//temp = matrix[tempy][sz - 1 - tempx];
			temp = matrix[tempy * (matrixSz)+(sz - 1 - tempx)];
			if (l > 0)
				matrix[tempy * (matrixSz)+(sz - 1 - tempx)] = tempPrev;
			else
				matrix[tempy * matrixSz + (sz - 1 - tempx)] = matrix[tempx * matrixSz + tempy];
			int oldTempx = tempx;
			tempx = tempy;
			tempy = sz - 1 - oldTempx;
			tempPrev = temp;
		}
	}


}

__global__ void rotateKernel(int matrixSz, int halfsz, int* matrix)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	int y_axis = ((tid + 1) / matrixSz);
	int sz = matrixSz;
	int x_axis = threadIdx.x;

	//if ()return;
	if (x_axis < sz - 1 - y_axis && x_axis >= y_axis && y_axis > halfsz)
	{
		int temp, tempPrev;
		int tempx = y_axis;
		int tempy = x_axis;
		for (int edgeSide = 0; edgeSide < 4; edgeSide++)
		{
			temp = matrix[tempy * (matrixSz)+(sz - 1 - tempx)];
			if (edgeSide > 0)
				matrix[tempy * (matrixSz)+(sz - 1 - tempx)] = tempPrev;
			else
				matrix[tempy * matrixSz + (sz - 1 - tempx)] = matrix[tempx * matrixSz + tempy];
			int oldTempx = tempx;
			tempx = tempy;
			tempy = sz - 1 - oldTempx;
			tempPrev = temp;
		}
	}


}

int main()
{
	int GivenMatrixSizeN = 15000;
	hipDeviceSynchronize();
	std::vector<std::vector<int>>matrix = std::vector<std::vector<int>>(GivenMatrixSizeN);
	int count = 0;
	//generate input matrix array data
	for (int i = 0; i < GivenMatrixSizeN; i++)
		for (int j = 0; j < GivenMatrixSizeN; j++)
			matrix[i].push_back(++count);
	int* oneDImage = new int[GivenMatrixSizeN * GivenMatrixSizeN];
	hipError_t cudaStatus = rotationCuda(oneDImage, matrix);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	//cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	return 1;
	//}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t rotationCuda(int* OneDMatrix, std::vector<std::vector<int>> matrix)
{
	int* dev_matrix = 0;
	int size = matrix.size() * matrix[0].size();
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}



	cudaStatus = hipMalloc((void**)&dev_matrix, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	int* dst = dev_matrix;
	auto t0 = clock();
	for (auto& vec : matrix)
	{
		auto sz = vec.size();
		cudaStatus = hipMemcpy(dst, &vec[0], vec.size() * sizeof(int), hipMemcpyHostToDevice);
		dst = dst + sz;
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}
	auto t1 = clock();


	int thdsize = ceil((double)matrix.size() / 2.0);
	int blockSize = 1;

	// Launch a kernel on the GPU
	
	//if (matrix.size() > 32)
	//{
	//	blockSize = matrix.size() / 32;
	//	thdsize = 32;

	//}
	//rotateKernel_badVersion << <blockSize, thdsize >> > (matrix.size(), ceil((double)size / 2.0), dev_matrix);

	if (matrix.size() * matrix.size() > 32)
	{
		blockSize = ceil((double)matrix.size() / 32.0);
		thdsize = 32;
	}
	rotateKernel << <blockSize, thdsize >> > (matrix.size(), ceil((double)size / 2.0), dev_matrix);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(OneDMatrix, dev_matrix, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;

	}
	auto t2 = clock();
	std::cout << t2 - t1 << " ms ellapsed\n"; //omit time evaluation of hipMalloc (can be reused after initialization)
	std::cout << t2 - t0 << " ms ellapsed(with hipMemcpy)\n";

#ifdef printMatrix
	for (int i = 0; i < matrix.size(); i++)
	{
		for (int j = 0; j < matrix.size(); j++)
		{
			std::cout << OneDMatrix[i * matrix.size() + j] << ",";
		}
		std::cout << std::endl;
	}
#endif

	system("pause");
Error:
	hipFree(dev_matrix);


	return cudaStatus;
}
